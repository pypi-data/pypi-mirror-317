#include "hip/hip_runtime.h"
// Adapted from https://github.com/vllm-project/vllm/blob/v0.6.5/csrc/moe/moe_align_sum_kernels.cu

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>
#include <torch/extension.h>

#include <THC/THCAtomics.cuh>

#ifdef USE_ROCM
#include <hip/hip_runtime.h>
#endif

#ifndef USE_ROCM
#define WARP_SIZE 32
#else
#define WARP_SIZE warpSize
#endif

#ifndef USE_ROCM
#define DevFuncAttribute_SET_MaxDynamicSharedMemorySize(FUNC, VAL) \
  hipFuncSetAttribute(reinterpret_cast<const void*>(FUNC), hipFuncAttributeMaxDynamicSharedMemorySize, VAL)
#else
#define DevFuncAttribute_SET_MaxDynamicSharedMemorySize(FUNC, VAL) \
  hipFuncSetAttribute(reinterpret_cast<const void*>(FUNC), hipFuncAttributeMaxDynamicSharedMemorySize, VAL)
#endif

#define CEILDIV(x, y) (((x) + (y)-1) / (y))

#define DISPATCH_CASE_INTEGRAL_TYPES(...)              \
  AT_DISPATCH_CASE(at::ScalarType::Byte, __VA_ARGS__)  \
  AT_DISPATCH_CASE(at::ScalarType::Char, __VA_ARGS__)  \
  AT_DISPATCH_CASE(at::ScalarType::Short, __VA_ARGS__) \
  AT_DISPATCH_CASE(at::ScalarType::Int, __VA_ARGS__)   \
  AT_DISPATCH_CASE(at::ScalarType::Long, __VA_ARGS__)

#define DISPATCH_INTEGRAL_TYPES(TYPE, NAME, ...) \
  AT_DISPATCH_SWITCH(TYPE, NAME, DISPATCH_CASE_INTEGRAL_TYPES(__VA_ARGS__))

__device__ __forceinline__ int32_t index(int32_t total_col, int32_t row, int32_t col) {
  // don't worry about overflow because num_experts is relatively small
  return row * total_col + col;
}

template <typename scalar_t>
__global__ void moe_align_block_size_kernel(scalar_t* __restrict__ topk_ids, int32_t* sorted_token_ids,
                                            int32_t* expert_ids, int32_t* total_tokens_post_pad, int32_t num_experts,
                                            int32_t block_size, size_t numel, int32_t* tokens_cnts, int32_t* cumsum) {
  const size_t tokens_per_thread = CEILDIV(numel, blockDim.x);
  const size_t start_idx = threadIdx.x * tokens_per_thread;

  for (int i = 0; i < num_experts; ++i) {
    tokens_cnts[index(num_experts, threadIdx.x + 1, i)] = 0;
  }

  /**
   * In the first step we compute token_cnts[thread_index + 1][expert_index],
   * which counts how many tokens in the token shard of thread_index are
   * assigned to expert expert_index.
   */
  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    ++tokens_cnts[index(num_experts, threadIdx.x + 1, topk_ids[i])];
  }

  __syncthreads();

  // For each expert we accumulate the token counts from the different threads.
  if (threadIdx.x < num_experts) {
    tokens_cnts[index(num_experts, 0, threadIdx.x)] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
      tokens_cnts[index(num_experts, i, threadIdx.x)] += tokens_cnts[index(num_experts, i - 1, threadIdx.x)];
    }
  }

  __syncthreads();

  // We accumulate the token counts of all experts in thread 0.
  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      cumsum[i] = cumsum[i - 1] + CEILDIV(tokens_cnts[index(num_experts, blockDim.x, i - 1)], block_size) * block_size;
    }
    *total_tokens_post_pad = cumsum[num_experts];
  }

  __syncthreads();

  /**
   * For each expert, each thread processes the tokens of the corresponding
   * blocks and stores the corresponding expert_id for each block.
   */
  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1]; i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }

  /**
   * Each thread processes a token shard, calculating the index of each token
   * after sorting by expert number. Given the example topk_ids =
   * [0,1,2,1,2,3,0,3,4] and block_size = 4, then the output would be [0, 6, *,
   * *, 1, 3, *, *, 2, 4, *, *, 5, 7, *, *, 8, *, *, *], where * represents a
   * padding value(preset in python).
   */
  for (int i = start_idx; i < numel && i < start_idx + tokens_per_thread; ++i) {
    int32_t expert_id = topk_ids[i];
    /** The cumsum[expert_id] stores the starting index of the tokens that the
     * expert with expert_id needs to process, and
     * tokens_cnts[threadIdx.x][expert_id] stores the indices of the tokens
     * processed by the expert with expert_id within the current thread's token
     * shard.
     */
    int32_t rank_post_pad = tokens_cnts[index(num_experts, threadIdx.x, expert_id)] + cumsum[expert_id];
    sorted_token_ids[rank_post_pad] = i;
    ++tokens_cnts[index(num_experts, threadIdx.x, expert_id)];
  }
}

void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts, int64_t block_size,
                          torch::Tensor sorted_token_ids, torch::Tensor experts_ids, torch::Tensor num_tokens_post_pad,
                          torch::Tensor token_cnts_buffer, torch::Tensor cumsum_buffer) {
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  DISPATCH_INTEGRAL_TYPES(topk_ids.scalar_type(), "moe_align_block_size_kernel", [&] {
    // calc needed amount of shared mem for `tokens_cnts` and `cumsum`
    // tensors
    const int32_t num_thread = max((int32_t)num_experts, WARP_SIZE);

    auto kernel = moe_align_block_size_kernel<scalar_t>;
    kernel<<<1, num_thread, 0, stream>>>(topk_ids.data_ptr<scalar_t>(), sorted_token_ids.data_ptr<int32_t>(),
                                         experts_ids.data_ptr<int32_t>(), num_tokens_post_pad.data_ptr<int32_t>(),
                                         num_experts, block_size, topk_ids.numel(),
                                         token_cnts_buffer.data_ptr<int32_t>(), cumsum_buffer.data_ptr<int32_t>());
  });
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("moe_align_block_size", &moe_align_block_size, "MOE Align Block Size (CUDA)");
}
